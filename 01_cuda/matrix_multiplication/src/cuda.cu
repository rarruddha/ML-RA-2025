#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "timer.h"

#include <stdio.h>
#include <iostream>
#include <exception>
#include <cmath>

using namespace std;

// Define a kernel function, which is the entry point
// for execution on the GPU
__global__ void matrix_multiplication(float *m1, float *m2, float *result, unsigned int m1_rows, unsigned int m1_cols, unsigned int m2_cols)
{
    // Get the row and column of the current element
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

    // Return if the current element is out of bounds
    if (i >= m1_rows || j >= m2_cols) {
        return;
    }

    // Compute the dot product of the row of m1 and the column of m2
    float value = 0;
    for (unsigned int k = 0; k < m1_cols; k++) {
        value += m1[i * m1_cols + k] * m2[k * m2_cols + j];
    }

    // Store the result in the output matrix
    result[i * m2_cols + j] = value;
}

vector<float> cuda_multiplication(const std::vector<float>& m1,
                                    const std::vector<float>& m2,
                                    unsigned int m1_rows,
                                    unsigned int m1_cols,
                                    unsigned int m2_cols)
{
    auto& timer = util::timers.gpu_add("CUDA Multiplication");
    // Allocate memory on the host
    vector<float> result(m1_rows * m2_cols);
    // Allocate memory on the device
    float *d_m1, *d_m2, *d_result;
    hipMalloc(&d_m1, m1_rows * m1_cols * sizeof(float));
    hipMalloc(&d_m2, m1_cols * m2_cols * sizeof(float));
    hipMalloc(&d_result, m1_rows * m2_cols * sizeof(float));
    // Copy data from host to device
    hipMemcpy(d_m1, m1.data(), m1_rows * m1_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m2, m2.data(), m1_cols * m2_cols * sizeof(float), hipMemcpyHostToDevice);
    // // sync cuda device
    // hipDeviceSynchronize();
    // Define grid and block size
    int n = 16;
    dim3 grid(ceil((float)m1_rows/n), ceil((float)m2_cols/n), 1);
    dim3 block(n, n, 1);
    // cout << "grid: " << grid.x << " " << grid.y << " " << grid.z << endl;
    // cout << "block: " << block.x << " " << block.y << " " << block.z << endl;
    // Launch kernel
    matrix_multiplication<<<grid, block>>>(d_m1, d_m2, d_result, m1_rows, m1_cols, m2_cols);
    // // sync cuda device
    // hipDeviceSynchronize();
    // Copy data from device to host
    hipMemcpy(result.data(), d_result, m1_rows * m2_cols * sizeof(float), hipMemcpyDeviceToHost);
    // Free memory on the device
    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_result);
    timer.stop();
    return result;
}